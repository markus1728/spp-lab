#include "hip/hip_runtime.h"
//
//  kernel.cu


#include <iostream>
#include <algorithm>
#include <cmath>
#include "ppm.h"

using namespace std;

/*********** Gray Scale Filter  *********/

/**
 * Converts a given 24bpp image into 8bpp grayscale using the GPU.
 */
__global__
void cuda_grayscale(int width, int height, BYTE *image, BYTE *image_out)
{
	int globalIdx = (gridDim.x * blockIdx.y + blockIdx.x) * (blockDim.x * blockDim.y) + (blockDim.x * threadIdx.y + threadIdx.x);
	image_out[globalIdx] = 0.0722f * image[globalIdx*3] + 0.7152f * image[globalIdx*3+1] + 0.2126 * image[globalIdx*3+2];
}


// 1D Gaussian kernel array values of a fixed size (make sure the number > filter size d)
__constant__ float cGaussian[64];
void cuda_updateGaussian(int r, double sd)
{
	
	float fGaussian[64];
	for (int i = 0; i < 2*r +1 ; i++)
	{
		float x = i - r;
		fGaussian[i] = expf(-(x*x) / (2 * sd*sd));
	}
	
	cout << "hipMemcpyToSymbol cGaussian Error: " << hipGetErrorString(hipMemcpyToSymbol(HIP_SYMBOL(cGaussian), fGaussian, 64*sizeof(float), 0, hipMemcpyHostToDevice)) << endl;
	
}
__device__
double cuda_gaussian(float x, double sigma)
{
	return expf(-(powf(x, 2)) / (2 * powf(sigma, 2)));
}

/*********** Bilateral Filter  *********/
// Parallel (GPU) Bilateral filter kernel
__global__ void cuda_bilateral_filter(BYTE* input, BYTE* output,
	int width, int height,
	int r, double sI, double sS)
{
	// Global Index
	int globalIdx = (gridDim.x * blockIdx.y + blockIdx.x) * (blockDim.x * blockDim.y) + (blockDim.x * threadIdx.y + threadIdx.x);
	unsigned char centrePx = input[globalIdx];

	int h = (int) (globalIdx / width);
	int w = globalIdx - (width * h);

	double iFiltered = 0;
	double wP = 0;

	for (int dy = -r; dy <= r; dy++)
	{
		int neighborY = h + dy;
		// falls ausserhalb des Bildes:
		if (neighborY < 0) neighborY = 0;
		else if (neighborY >= height) neighborY = height - 1;
		
		for (int dx = -r; dx <= r; dx++)
		{
			int neighborX = w+dx;
			// falls ausserhalb des Bildes:
			if (neighborX < 0) neighborX = 0;
			else if (neighborX >= width) neighborX = width - 1;

			unsigned char curPx = input[neighborY*width+neighborX];
			
			double w = (cGaussian[dy+r] * cGaussian[dx+r]) * cuda_gaussian(centrePx - curPx, sI);
			
			iFiltered += w * curPx;
			wP += w;
		}
	}
	output[globalIdx] = iFiltered / wP;
}


void gpu_pipeline(const Image & input, Image & output, int r, double sI, double sS)
{
	// Events to calculate gpu run time
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// GPU related variables
	BYTE *d_input = NULL;
	BYTE *d_image_out[2] = {0}; //temporary output buffers on gpu device
	int image_size = input.cols*input.rows;
	int suggested_blockSize;   // The launch configurator returned block size 
	int suggested_minGridSize; // The minimum grid size needed to achieve the maximum occupancy for a full device launch

	// ******* Grayscale kernel launch *************

	//Creating the block size for grayscaling kernel
	cout << "hipOccupancyMaxPotentialBlockSize Error: " << hipGetErrorString(hipOccupancyMaxPotentialBlockSize( &suggested_minGridSize, &suggested_blockSize, cuda_grayscale)) << endl;
	cout << "Suggested block size: " << suggested_blockSize << " Suggested min grid size: " << suggested_minGridSize << endl;

	int block_dim_x, block_dim_y;
	block_dim_x = block_dim_y = (int) sqrt(suggested_blockSize); 

	dim3 gray_block(block_dim_x, block_dim_y); // 2 pts

	int actual_blockSize = block_dim_x * block_dim_y;
	int actual_gridSize = round(image_size / (float)actual_blockSize + 0.5);

	int grid_dim_x, grid_dim_y;
	grid_dim_x = grid_dim_y = round(sqrt(actual_gridSize)+0.5);
	dim3 gray_grid(grid_dim_x, grid_dim_y);

	cout << "Actual block size: " << actual_blockSize << " Actual grid size: " << actual_gridSize << endl;
	
	// Allocate the intermediate image buffers for each step
	Image img_out(input.cols, input.rows, 1, "P5");
	for (int i = 0; i < 2; i++)
	{  
		cout << "hipMallocManaged d_img_out Error: " << hipGetErrorString(hipMallocManaged(&d_image_out[i], image_size*sizeof(BYTE))) << endl;
		cout << "hipMemset d_img_out Error: " << hipGetErrorString(hipMemset(d_image_out[i], 0, image_size*sizeof(BYTE))) << endl;
	}

	cout << "hipMallocManaged d_input Error: " << hipGetErrorString(hipMallocManaged((void **)&d_input, image_size*3*sizeof(BYTE))) << endl;
	cout << "hipMemcpy d_input Error: " << hipGetErrorString(hipMemcpy(d_input, input.pixels, image_size*3*sizeof(BYTE), hipMemcpyHostToDevice)) << endl;

	hipEventRecord(start, 0); // start timer
	
	// Convert input image to grayscale
	cout << "Call 'cuda_grayscale' with gray_grid " << gray_grid.x << " x " << gray_grid.y << " x " << gray_grid.z << endl << "and gray_block " << gray_block.x << " x " << gray_block.y << " x " << gray_block.z << endl << "Image size: " << image_size << endl;
	cuda_grayscale<<<gray_grid, gray_block>>>(input.cols, input.rows, d_input, d_image_out[0]);

	hipEventRecord(stop, 0); // stop timer
	hipEventSynchronize(stop);

        // Calculate and print kernel run time
	hipEventElapsedTime(&time, start, stop);
	cout << "GPU Grayscaling time: " << time << " (ms)\n";
	cout << "Launched blocks of size " << gray_block.x * gray_block.y << endl;
    
	cout << "hipMemcpy d_image_out Error: " << hipGetErrorString(hipMemcpy(img_out.pixels, d_image_out[0], image_size*sizeof(BYTE), hipMemcpyDeviceToHost)) << endl;
	savePPM(img_out, "image_gpu_gray.ppm");
	

	// ******* Bilateral filter kernel launch *************
	
	//Creating the block size for grayscaling kernel
	cout << "hipOccupancyMaxPotentialBlockSize Error: " << hipGetErrorString(hipOccupancyMaxPotentialBlockSize( &suggested_minGridSize, &suggested_blockSize, cuda_bilateral_filter)) << endl; 
        cout << "Suggested block size: " << suggested_blockSize << " Suggested min grid size: " << suggested_minGridSize << endl;

        block_dim_x = block_dim_y = (int) sqrt(suggested_blockSize); 

        dim3 bilateral_block(block_dim_x, block_dim_y); // 2 pts

	actual_blockSize = block_dim_x * block_dim_y;
	actual_gridSize = round(image_size / (float)actual_blockSize + 0.5);

	grid_dim_x = grid_dim_y = round(sqrt(actual_gridSize)+0.5);
	dim3 bilateral_grid(grid_dim_x, grid_dim_y);

	cout << "Actual block size: " << actual_blockSize << " Actual grid size: " << actual_gridSize << endl;

        // Create gaussain 1d array
        cuda_updateGaussian(r,sS);

        hipEventRecord(start, 0); // start timer
	
	cuda_bilateral_filter<<<bilateral_grid, bilateral_block>>>(d_image_out[0], d_image_out[1], input.cols, input.rows, r, sI, sS);

        hipEventRecord(stop, 0); // stop timer
        hipEventSynchronize(stop);

        // Calculate and print kernel run time
        hipEventElapsedTime(&time, start, stop);
        cout << "GPU Bilateral Filter time: " << time << " (ms)\n";
        cout << "Launched blocks of size " << bilateral_block.x * bilateral_block.y << endl;

        // Copy output from device to host
	cout << "hipMemcpy d_image_out Error: " << hipGetErrorString(hipMemcpy(output.pixels, d_image_out[1], image_size*sizeof(BYTE), hipMemcpyDeviceToHost)) << endl;
	
	
        // ************** Finalization, cleaning up ************

        hipFree(d_image_out);
	hipFree(d_input);
}
